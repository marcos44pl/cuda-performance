#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#include "ImageManager.h"
#include "headers.h"
#include "Timer.h"
#include "test_cases.h"
#include "utils.h"
#include "computation.h"

std::map<std::string,execKernel> kernel_map =
	{
		{"rotation_global_mem", &rotation_global_mem},
		{"rotation_shared_mem", &rotation_shared_mem},
		{"sobel_filter_coalesc", &sobel_filter_coalesc},
		{"sobel_filter_non_coalesc", &sobel_filter_non_coalesc}
	};

Fraction* execDevice(StartArgs args);

StartArgs parsInputArguments(const int argc, char *argv[])
{
	StartArgs args;

	//default simulation settings
	args.NUM_OF_ITERATIONS = 10;
	args.X_SIZE = 100;
	args.Y_SIZE = 100;
	args.Z_SIZE = 100;
	args.type = deviceSimulationType::GLOBAL;
	args.print = false;
	return args;
}

int main(int argc, char *argv[])
{
	initCuda();
	printf("Testing UM optimalizations\n");
	//testFl16FullyConnectedFwdCudaNN();
	//testFl16Cudnn();
	//testFl16PoolCudaNN();
	//testFl16ConvCudaNN();
	//testFluidSimStd();
	//testFluidSimUM();
	//testFluidSimUM(false);
	ImageManager image;
	image.createEmpty(30000,30000);
	for(int i = 0; i < 10;++i)
	{
		for(auto const& pair : kernel_map)
		{
			testCudaMemGeneric(image,pair.first + std::string(" UM std "),
								createUMem,
								pair.second,
								copyMock,
								freeUM);
			testCudaMemGeneric(image,pair.first + std::string(" UM opt "),
								createUMemOpt,
								pair.second,
								copyMock,
								freeUM);
			testCudaMemGeneric(image,pair.first + std::string(" MemCpy std "),
								createStdMem,
								pair.second,
								copyStdMemBack,
								freeStd);
		}
	}
	image.clear();
	for(int i = 0; i < 10;++i)
	{
		testSobelOversubUMOpt();
		testSobelOversubStd();
		testSobelOversubUM();
		testSobelOversubMultiImgStd();
		testSobelStreamUM(false);
		testSobelStreamUM(true);
		testSobelStreamStd();
	}

	Timer::getInstance().printResults();
	hipDeviceSynchronize();
    hipProfilerStop();
    hipDeviceReset();
    printf("end\n");

	return 0;
}
