#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#include "ImageManager.h"
#include "headers.h"
#include "Timer.h"
#include "test_cases.h"
#include "utils.h"
#include "computation.h"

std::map<std::string,execKernel> kernel_map =
	{
		{"rotation_global_mem", &rotation_global_mem},
		{"rotation_shared_mem", &rotation_shared_mem},
		{"sobel_filter_coalesc", &sobel_filter_coalesc},
		{"sobel_filter_non_coalesc", &sobel_filter_non_coalesc}
	};

Fraction* execDevice(StartArgs args);

StartArgs parsInputArguments(const int argc, char *argv[])
{
	StartArgs args;

	//default simulation settings
	args.NUM_OF_ITERATIONS = 10;
	args.X_SIZE = 100;
	args.Y_SIZE = 100;
	args.Z_SIZE = 100;
	args.type = deviceSimulationType::GLOBAL;
	args.print = false;
	return args;
}

int main(int argc, char *argv[])
{
	initCuda();
	/*ImageManager image;
	image.createEmpty(20000,20000);
	printf("Testing UM optimalizations\n");

	for(auto const& pair : kernel_map)
	{
		testCudaMemGeneric(image,pair.first + std::string(" UM std "),
							createUMem,
							pair.second,
							copyMock,
							freeUM);
		testCudaMemGeneric(image,pair.first + std::string(" UM opt "),
							createUMemOpt,
							pair.second,
							copyMock,
							freeUM);
		testCudaMemGeneric(image,pair.first + std::string(" MemCpy std "),
							createStdMem,
							pair.second,
							copyStdMemBack,
							freeStd);
	}
	image.clear();

	testSobelOversubStd();
	testSobelOversubUM();
	testFluidSimStd();
	testFluidSimUM();
	testFluidSimUM(false);
	testSobelStreamUM(false);
	testSobelStreamUM(true);
	testSobelStreamStd();
	testFl16Cudnn();*/
	testFl16ConvCudaNN();
	Timer::getInstance().printResults();
    hipProfilerStop();
    hipDeviceReset();
    printf("end\n");

	return 0;
}
