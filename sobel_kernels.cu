#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "config.h"
#include "headers.h"



__device__ uchar compute_sobel(uchar tl, //top left
							   uchar tm, //top middle
							   uchar tr, //top right
							   uchar ml, //middle left
							   uchar mr, //middle right
							   uchar bl, //bottom left
							   uchar bm, //bottom middle
							   uchar br, //bottom right
							   float norm)
{
	short horz = tl + 2 * tm + tr - br - 2 * bm - bl;
	short vert = tl + 2 * ml + bl - tr - 2 * mr - br;
	short all = (short)(norm * float(fabsf(horz) + fabsf(vert)));

	if(all < 0)
		all = 0;
	if(all > 0xff)
		all = 0xff;

	return all;
}

__device__ void read_write_filter_block(int x, int y,uchar *in, uchar *out,uint width,float norm)
{
	uchar value = compute_sobel(in[PXL_ID(x,y,width)],
							  in[PXL_ID(x,y+1,width)],
							  in[PXL_ID(x,y+2,width)],
							  in[PXL_ID(x+1,y,width)],
							  in[PXL_ID(x+1,y+2,width)],
							  in[PXL_ID(x+2,y,width)],
							  in[PXL_ID(x+2,y+1,width)],
							  in[PXL_ID(x+2,y+2,width)],
							  norm);
	out[PXL_ID(x+1,y+1,width)] = value;
}

__global__ void kernel_sobel_filter_coalesc(uchar *in, uchar *out,uint width, uint height,uint pxl_p_thd,float norm)
{
	int x = blockIdx.x*BATCH_W + threadIdx.x - RAD;
	int y = blockIdx.y*BATCH_H + threadIdx.y - RAD;

	if(x >= 0 && y >= 0 && y < height)
	{
		for(int ix = x; ix < width; ix += blockDim.x)
		{
			read_write_filter_block(ix,y,in,out,width,norm);
		}
	}
}

__global__ void kernel_sobel_filter_non_coalesc(uchar *in, uchar *out,uint width, uint height,uint pxl_p_thd,float norm)
{
	int x = blockIdx.x*BATCH_W + threadIdx.x - RAD;
	int y = blockIdx.y*BATCH_H + threadIdx.y - RAD;

	if(x >= 0 && y >= 0 && y < height)
	{
		for(int ix =  x * pxl_p_thd;  ix < (x+1) * pxl_p_thd; ++ix)
		{
			if(ix < width)
			{
				read_write_filter_block(ix,y,in,out,width,norm);
			}
		}
	}
}

void sobel_filter_coalesc(uchar* in, uchar* out,size_t width, size_t height)
{
	exec_kernel(in,out,width,height,kernel_sobel_filter_coalesc);
}

void sobel_filter_non_coalesc(uchar* in, uchar* out,size_t width, size_t height)
{
	exec_kernel(in,out,width,height,kernel_sobel_filter_non_coalesc);
}

