#include "hip/hip_runtime.h"
#include <cmath>
#include <stdlib.h>
#include <stdio.h>

#include "ImageManager.h"
#include "Timer.h"
#include "headers.h"
#include "config.h"
#include "utils.h"

#define IMG_SIZE_GB 6
#define IMG_NUM 200
#define IMG_W 1920
#define IMG_H 1080

StartArgs parsInputArguments()
{
	StartArgs args;

	//default simulation settings
	args.NUM_OF_ITERATIONS = 100;
	args.X_SIZE = 200;
	args.Y_SIZE = 200;
	args.Z_SIZE = 200;
	args.type = deviceSimulationType::SHARED_3D_LAYER;
	args.print = false;
	return args;
}

void testCudaMemGeneric(ImageManager& 		image,
						 std::string const& name,
						 createMemFunc      beforeFunc,
						 execKernel 		kernelFunc,
						 copyMemAfterFunc	afterFunc,
						 freeMem 			freeFunc)
{
	printf("%s\n",name.c_str());
	uchar* in,*out,*h_out;
	auto size = image.get_size();
	freeMemory();
	Timer::getInstance().start(name);
	in = beforeFunc(image.get_data(),size);
	out = in + size;
	kernelFunc(in,out,image.get_width(),image.get_height());
	hipDeviceSynchronize();
	cudaCheckError();
	h_out = afterFunc(out,size);
	Timer::getInstance().stop(name);
	//image.save("result.jpg",h_out);
	freeFunc(in,out,h_out);
}

void testStreamImgProcessingStd(kernelPtr kernel)
{
	std::string name = "Stream Image Processing Std";
    Timer::getInstance().start(name);
	const uint size = IMG_H * IMG_W * 3;
	const uint sizePerCh = IMG_H * IMG_W;
	const uint sizePerChBytes = IMG_H * IMG_W * sizeof(uchar);
	const int streamNum = 3;
	const ulong totalSize = sizeof(uchar) * IMG_NUM * size;
	hipStream_t streams[streamNum];
	uchar* d_mem = createStdMem(nullptr,totalSize);
	uchar* h_mem;
	hipHostMalloc((void**)&h_mem,totalSize);
	memset(h_mem, 0, totalSize);
	uchar* c_res_mem = d_mem + size * IMG_NUM;

	for(int i = 0; i <streamNum; ++i)
	{
		hipStreamCreate(&streams[i]);
		cudaCheckError();
	}

	dim3 thsPerBlck(BATCH_W,BATCH_H);
	dim3 blckNum(IMG_W/(BATCH_W * PXL_PER_THD) + 1,IMG_H/(BATCH_H) + 1);
	int totalChCount = IMG_NUM*3;
	for(int i =0; i< totalChCount;++i)
	{
		int s_id = i % streamNum;
		 hipMemcpyAsync(&d_mem[i], &h_mem[i],sizePerChBytes,
		                 hipMemcpyHostToDevice, streams[s_id]);
		cudaCheckError();
	}


	for(int i =0; i< totalChCount;++i)
	{
		int s_id = i % streamNum;
		kernel<<<blckNum, thsPerBlck, 0, streams[s_id]>>>(&d_mem[i * sizePerCh],&c_res_mem[i * sizePerCh],
														  IMG_W,IMG_H,PXL_PER_THD,IMAGE_SCALE);
		cudaCheckError();
	}

	for(int i =0; i< totalChCount;++i)
	{
		int s_id = i % streamNum;
		 hipMemcpyAsync(&h_mem[i], &d_mem[i],sizePerChBytes,
				 	 	 hipMemcpyDeviceToHost, streams[s_id]);
		cudaCheckError();
	}
	hipDeviceSynchronize();
	cudaCheckError();
	testRead(h_mem,totalSize);
    Timer::getInstance().stop(name);
	hipFree(d_mem);
	cudaCheckError();
	hipHostFree(h_mem);
	cudaCheckError();
}

void testStreamImgProcessingUm(kernelPtr kernel,std::string name,int imgCount,bool withAdvise)
{
	if(withAdvise)
		name += " Opt";
    Timer::getInstance().start(name);

	const ulong size = IMG_H * IMG_W * 3;
	const ulong totalSize = sizeof(uchar) * imgCount * size;
	const uint totalChCount = imgCount*3;
	const uint sizePerCh = IMG_H * IMG_W;
	const uint sizePerChBytes = sizeof(uchar) * sizePerCh;

	const int streamNum = 5;
	uchar* umem = createUMem(nullptr,totalSize);
	uchar* res_umem = umem + size * imgCount;
	memset(umem, 0, totalSize);
	hipStream_t streams[streamNum];
	hipEvent_t events[streamNum];
	for(int i = 0; i <streamNum; ++i)
	{
		hipStreamCreate(&streams[i]);
		cudaCheckError();
		hipEventCreate(&events[i]);
		cudaCheckError();
	}

	dim3 thsPerBlck(BATCH_W,BATCH_H);
	dim3 blckNum(IMG_W/(BATCH_W * PXL_PER_THD) + 1,IMG_H/(BATCH_H) + 1);

	int device =-1;
	hipGetDevice(&device);
	cudaCheckError();
	if(withAdvise)
	{
		hipMemPrefetchAsync(umem,sizePerChBytes, device,streams[1]);
		cudaCheckError();
		hipEventRecord(events[0], streams[1]);
		cudaCheckError();
	}
	for(int i =0; i< totalChCount;++i)
	{
		int s_id = i % streamNum;
		int s1_id = (i + 1) % streamNum;
		hipEventSynchronize(events[s_id]);
		hipEventSynchronize(events[s1_id]);
		kernel<<<blckNum, thsPerBlck, 0, streams[s_id]>>>(umem + i * sizePerCh,res_umem + i * sizePerCh,
														  IMG_W,IMG_H,PXL_PER_THD,IMAGE_SCALE);
		hipEventRecord(events[s_id], streams[s_id]);

		if(withAdvise)
		{
			if(i < totalChCount -1)
			{
				hipStreamSynchronize(streams[s1_id]);
				hipMemPrefetchAsync(umem + (i+1) * sizePerCh, sizePerChBytes, device, streams[s1_id]);
				hipEventRecord(events[s1_id], streams[s1_id]);
			}
			hipMemPrefetchAsync(res_umem + i * sizePerCh, sizePerChBytes, hipCpuDeviceId, streams[s_id]);
		}
	}
	hipDeviceSynchronize();
	cudaCheckError();
	testRead(res_umem,totalSize);
    Timer::getInstance().stop(name);
	hipFree(umem);
}

void testOversubStd(kernelPtr kernel)
{
	const ulong kb = 1024;
	const ulong imgSize = kb * kb * kb * IMG_SIZE_GB;
	const uint channelNum = 3;
	const std::string name = "Oversubscription standard mem with overlapping";
	const double freeMemLeave = 0.05;
	const uint streamNum = 10;

	Timer::getInstance().start(name);

	uint width = sqrt(imgSize / (channelNum * sizeof(uchar)));
	uint height = width;
	ulong realSizeCh = width * height * sizeof(uchar);
	ulong realSize = realSizeCh * channelNum;
	auto freeMem = freeMemory();
	ulong sizetoDevMAlloc = ulong((double)freeMem * (1. - freeMemLeave));
	uint minStreamsNum = ceil((double)(realSize * 2) / sizetoDevMAlloc);
	uint realStreamNum = max(streamNum,minStreamsNum);
	ulong streamSize = realSize / realStreamNum;
	int streamHeight = streamSize / (width * sizeof(uchar));

	uchar* pitchedHMem,*dev_mem;
	hipHostMalloc((void**)&pitchedHMem,realSize);
	cudaCheckError();
	hipMalloc((void**)&dev_mem,sizetoDevMAlloc);
	cudaCheckError();
	memset(pitchedHMem, 0, realSize);
	realStreamNum += 2;
	hipStream_t* streams = new hipStream_t[realStreamNum];
	hipEvent_t* cpyEvents = new hipEvent_t[realStreamNum];
	int h_prior;
	hipDeviceGetStreamPriorityRange(nullptr,&h_prior);
	for(int i = 0; i <realStreamNum; ++i)
	{
		hipStreamCreateWithPriority(&streams[i],hipStreamDefault,h_prior++);
		cudaCheckError();
		hipEventCreate(&cpyEvents[i]);
		cudaCheckError();
	}
	dim3 thsPerBlck(BATCH_W,BATCH_H);

	uchar* d_ptr = dev_mem;
	uchar* h_ptr = pitchedHMem;
	int streamsEnded = 0;
	for(int i = 0,ch_i = 0; i <realStreamNum; ++i,++ch_i)
	{
		ulong offsetH = ch_i * streamSize;
		ulong curStremSize = streamSize;
		uint curHeight = streamHeight;
		if(offsetH > realSizeCh)
		{
			ch_i = 0;
			d_ptr += realSizeCh;
			h_ptr += realSizeCh;
			offsetH = 0;
		}
		if(offsetH + streamSize > realSizeCh)
		{
			curStremSize = realSizeCh - offsetH;
			curHeight = curStremSize / (width * sizeof(uchar));
		}
		ulong offsetD = offsetH * 2;

		if(d_ptr + offsetD + 2 *  curStremSize > dev_mem + sizetoDevMAlloc) // if we exceeded gpu mem we back to the begin
		{
			d_ptr = dev_mem;
			offsetD = 0;
			for(int j = streamsEnded; j <= i; ++j)
			{
				hipEventSynchronize(cpyEvents[j]); // we have to wait for the streams to get memory
				cudaCheckError();
				streamsEnded++;
			}
		}

		dim3 blckNum(width/(BATCH_W * PXL_PER_THD) + 1,curHeight/(BATCH_H) + 1);


		/*printf("Stream%d  Host off: %d Dev off: %p curHeight: %d curStreamS: %d\n",i,
								(offsetH)/(kb*kb),
								(void *)(d_ptr +offsetD),
								curHeight,
								curStremSize/(kb*kb));*/

		hipMemcpyAsync(&d_ptr[offsetD], &h_ptr[offsetH], curStremSize, hipMemcpyHostToDevice, streams[i]);
		cudaCheckError();
		kernel<<<blckNum, thsPerBlck, 0, streams[i]>>>(&d_ptr[offsetD],&d_ptr[offsetD + curStremSize],width,curHeight,PXL_PER_THD,IMAGE_SCALE);
		cudaCheckError();
		hipMemcpyAsync(&h_ptr[offsetH], &d_ptr[offsetD + curStremSize], curStremSize, hipMemcpyDeviceToHost, streams[i]);
		hipEventRecord(cpyEvents[i], streams[i]);
		cudaCheckError();
	}
	hipDeviceSynchronize();
	for(int i = 0; i <realStreamNum; ++i)
	{
		hipStreamDestroy(streams[i]); // we have to free the stream resources for the others
		cudaCheckError();
		hipEventDestroy(cpyEvents[i]);
		cudaCheckError();
	}
	testRead(pitchedHMem,realSize);

	hipFree(dev_mem);
	cudaCheckError();
	hipHostFree(pitchedHMem);
	cudaCheckError();
	Timer::getInstance().stop(name);
}

void testOversubNaiveUM(kernelPtr kernel,bool withAdvise /*=true*/)
{
	const ulong kb = 1024;
	const ulong imgSize = kb * kb * kb * IMG_SIZE_GB;
	const uint channelNum = 3;
	uint width = sqrt(imgSize / (channelNum * sizeof(uchar)));
	uint height = width;
	ulong realSizeCh = width * height * sizeof(uchar);
	ulong realSize = realSizeCh * channelNum;
	std::string name = "Oversubscription unified memory naive";
	if(withAdvise)
		name += " advised";
	Timer::getInstance().start(name);
	int device =-1;
	hipGetDevice(&device);
	uchar* um_data;
	if(withAdvise)
		um_data = createUMemOpt(nullptr,realSize);
	else
		um_data = createUMem(nullptr,realSize);

	exec_kernel(um_data,um_data + realSize,width,height,kernel);
	hipDeviceSynchronize();
	cudaCheckError();
	if(withAdvise)
	{
		hipMemPrefetchAsync(um_data + realSize,realSize,hipCpuDeviceId,NULL);
		cudaCheckError();
	}
	testRead(um_data + realSize,realSize);

	hipFree(um_data);
	cudaCheckError();
	Timer::getInstance().stop(name);
}

void testOversubMultiImgStd(kernelPtr kernel)
{
	const std::string name = "Oversubscription multi_img std with overlapping";
	Timer::getInstance().start(name);

	const ulong kb = 1024;
	const ulong totalSize = kb * kb * kb * IMG_SIZE_GB;
	const uint channelNum = 3;
	const uint chSize = IMG_W * IMG_H;
	const uint chSizeBytes = chSize * sizeof(uchar);
	const uint size = chSize * channelNum;
	auto freeMem = freeMemory();
	float freeMemLeave = 0.05;
	ulong sizetoDevMAlloc = ulong((double)freeMem * (1. - freeMemLeave));
	uint imgCount = totalSize / (size);
	printf("ImgCount: %d\n",imgCount);
	uint chCount = imgCount * channelNum;
	uchar* d_mem,*d_result_mem,*h_mem,*h_result_mem;
	hipMalloc(&d_mem,sizetoDevMAlloc);
	cudaCheckError();
	hipHostMalloc((void**)&h_mem,totalSize*2);
	cudaCheckError();
	memset(h_mem, 0, totalSize);
	h_result_mem = h_mem + size * imgCount;
	d_result_mem = d_mem + size * imgCount;
	uint chOnGpuCount = uint(sizetoDevMAlloc/ chSizeBytes);
	uint streamNum = 5;
	uint loopCount = chOnGpuCount / streamNum;

	hipStream_t* streams = new hipStream_t[streamNum];
	hipEvent_t* cpyEvents = new hipEvent_t[streamNum];
	int h_prior;
	hipDeviceGetStreamPriorityRange(nullptr,&h_prior);
	cudaCheckError();

	for(int i = 0; i <streamNum; ++i)
	{
		hipStreamCreateWithPriority(&streams[i],hipStreamDefault,h_prior++);
		cudaCheckError();
		hipEventCreate(&cpyEvents[i]);
		cudaCheckError();
	}

	dim3 thsPerBlck(BATCH_W,BATCH_H);
	dim3 blckNum(IMG_W/(BATCH_W * PXL_PER_THD) + 1,IMG_H/(BATCH_H) + 1);


	for(int i = 0; i < loopCount;++i)
	{
		int batch_num_i = min(streamNum,chCount - i * loopCount);
		for(int j =0; j< batch_num_i;++j)
		{
			hipMemcpyAsync(&d_mem[j * chSize], &h_mem[j * chSize + i * streamNum * chSize],chSizeBytes,
			                 hipMemcpyHostToDevice, streams[j]);
			cudaCheckError();
		}


		for(int j =0; j< batch_num_i;++j)
		{
			kernel<<<blckNum, thsPerBlck, 0, streams[j]>>>(&d_mem[i * chSize],
														   &d_result_mem[i * chSize],
															  IMG_W,IMG_H,PXL_PER_THD,IMAGE_SCALE);
			cudaCheckError();
		}

		for(int j =0; j< batch_num_i;++j)
		{
			 hipMemcpyAsync(&h_result_mem[j * chSize + i * streamNum * chSize],
					         &d_result_mem[j * chSize],chSizeBytes,
					 	 	 hipMemcpyDeviceToHost, streams[j]);
			cudaCheckError();
		}
	}

	hipDeviceSynchronize();
	cudaCheckError();
	testRead(h_result_mem,totalSize/2);
    Timer::getInstance().stop(name);
	hipFree(d_mem);
	cudaCheckError();
	hipHostFree(h_mem);
	cudaCheckError();
}

void testOversubUMOpt(kernelPtr kernel,bool advised)
{
	const ulong kb = 1024;
	const ulong totalSize = kb * kb * kb * IMG_SIZE_GB;
	const uint channelNum = 3;
	uint width = IMG_W;
	uint height = IMG_H;
	uint imgCount = totalSize / (width * height * channelNum * sizeof(uchar));
	printf("ImgCount: %d\n",imgCount);
	std::string name = "Oversubscription multi_img UM streams ";
	testStreamImgProcessingUm(kernel,name + std::to_string(advised),imgCount,advised);
}

void testFluidSimStd()
{
    Timer::getInstance().start("Fluid simulation std mem");
    StartArgs args = parsInputArguments();
    Fraction* space = initSpace(args,false,false);
    FluidParams *d_params, params = initParams();

    hipMalloc((void **)&d_params, sizeof(FluidParams));
    hipMemcpy(d_params, &params, sizeof(FluidParams), hipMemcpyHostToDevice);

    if (NULL == space)
        exit(-1);

    Fraction *d_space, *d_result;
    int totalSize = sizeof(Fraction)*args.SIZE(), i;
    void *result = new Fraction[totalSize];

    hipMalloc((void **)&d_space, totalSize);
    hipMalloc((void **)&d_result, totalSize);
    cudaCheckErrors("Mallocs");
    hipMemcpy(d_space, space, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, totalSize, hipMemcpyHostToDevice);

    cudaCheckErrors("Copy mem");
    printf("StdMem Simulation started\n");

    for (i = 0; i<args.NUM_OF_ITERATIONS; ++i)
    {
        simulation(args, d_params, d_space, d_result);
        swapPointers(d_space, d_result);
        hipDeviceSynchronize();

    	if (i % 2 == 0)
    		hipMemcpy(space, d_space, totalSize, hipMemcpyDeviceToHost);
    	else
    		hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);

    	testRead(space,totalSize);
    }


    Timer::getInstance().stop("Fluid simulation std mem");
    printf("Simulation completed\n");
    free(space);
    hipFree(d_params);
    hipFree(d_space);
    hipFree(d_result);
    cudaCheckErrors("Free mem");
}

void testFluidSimUM(bool withAdvise)
{
    std::string name = withAdvise ? "Fluid simulation UM advised" : "Fluid simulation UM std";
    Timer::getInstance().start(name);

	int device = -1;
	hipGetDevice(&device);
    StartArgs args = parsInputArguments();
    int totalSize = sizeof(Fraction)*args.SIZE(), i;
    FluidParams *um_params, params = initParams();
    hipMallocManaged(&um_params, sizeof(FluidParams));
    memcpy(um_params,&params,sizeof(FluidParams));
    Fraction *buffer,*result,*start;
    hipMallocManaged(&buffer,totalSize);
    cudaCheckErrors("UM Mallocs");
    if(withAdvise)
    {
    	printf("UM advised Simulation started\n");
    	hipMemAdvise(um_params,sizeof(FluidParams),hipMemAdviseSetReadMostly,device);
    	cudaCheckError();
    	hipMemPrefetchAsync(um_params,sizeof(FluidParams),device,NULL);
    	cudaCheckError();
    	hipMemPrefetchAsync(buffer,totalSize,device,NULL);
    	cudaCheckError();
    }
    else
    {
    	printf("UM Simulation started\n");
    }
    Fraction* space = initSpace(args,true,withAdvise,device);
    for (i = 0; i<args.NUM_OF_ITERATIONS; ++i)
    {
        simulation(args, um_params, space, buffer);
        swapPointers(space, buffer);
        hipDeviceSynchronize();

    	result = (i % 2 == 0) ? space : buffer;
    	start = (i % 2 == 0)  ? buffer : space;

        if(withAdvise)
        {
        	hipMemPrefetchAsync(result,totalSize,hipCpuDeviceId,NULL);
        	cudaCheckError();
        	hipMemPrefetchAsync(start,totalSize,device,NULL);
        	cudaCheckError();
        }

    	testRead(result,totalSize);

        if(withAdvise)
        {
        	hipMemPrefetchAsync(result,totalSize,device,NULL);
        	cudaCheckError();
        }
    }

    Timer::getInstance().stop(name);
    printf("Simulation completed\n");

    hipFree(space);
    hipFree(buffer);
    hipFree(um_params);
}

