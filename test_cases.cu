#include "hip/hip_runtime.h"
#include <cmath>
#include <stdlib.h>
#include <stdio.h>

#include "ImageManager.h"
#include "Timer.h"
#include "headers.h"
#include "config.h"
#include "utils.h"

#define IMG_SIZE_GB 4
#define IMG_NUM 200
#define IMG_W 1920
#define IMG_H 1080

StartArgs parsInputArguments()
{
	StartArgs args;

	//default simulation settings
	args.NUM_OF_ITERATIONS = 100;
	args.X_SIZE = 200;
	args.Y_SIZE = 200;
	args.Z_SIZE = 200;
	args.type = deviceSimulationType::SHARED_3D_LAYER;
	args.print = false;
	return args;
}

void testCudaMemGeneric(ImageManager& 		image,
						 std::string const& name,
						 createMemFunc      beforeFunc,
						 execKernel 		kernelFunc,
						 copyMemAfterFunc	afterFunc,
						 freeMem 			freeFunc)
{
	printf("%s\n",name.c_str());
	uchar* in,*out,*h_out;
	auto size = image.get_size();
	freeMemory();
	Timer::getInstance().start(name);
	in = beforeFunc(image.get_data(),size);
	out = in + size;
	kernelFunc(in,out,image.get_width(),image.get_height());
	hipDeviceSynchronize();
	cudaCheckError();
	h_out = afterFunc(out,size);
	Timer::getInstance().stop(name);
	//image.save("result.jpg",h_out);
	freeFunc(in,out,h_out);
}

void testOversubStd(kernelPtr kernel)
{
	const ulong kb = 1024;
	const ulong imgSize = kb * kb * kb * IMG_SIZE_GB;
	const uint channelNum = 3;
	const std::string name = "Oversubscription standard mem with overlapping";
	const double freeMemLeave = 0.05;
	const uint streamNum = 10;

	Timer::getInstance().start(name);

	uint width = sqrt(imgSize / (channelNum * sizeof(uchar)));
	uint height = width;
	ulong realSizeCh = width * height * sizeof(uchar);
	ulong realSize = realSizeCh * channelNum;
	auto freeMem = freeMemory();
	ulong sizetoDevMAlloc = ulong((double)freeMem * (1. - freeMemLeave));
	uint minStreamsNum = ceil((double)(realSize * 2) / sizetoDevMAlloc);
	uint realStreamNum = max(streamNum,minStreamsNum);
	ulong streamSize = realSize / realStreamNum;
	int streamHeight = streamSize / (width * sizeof(uchar));

	uchar* pitchedHMem,*dev_mem;
	hipHostMalloc((void**)&pitchedHMem,realSize);
	cudaCheckError();
	hipMalloc((void**)&dev_mem,sizetoDevMAlloc);
	cudaCheckError();
	memset(pitchedHMem, 0, realSize);
	realStreamNum += 2;
	hipStream_t* streams = new hipStream_t[realStreamNum];
	hipEvent_t* cpyEvents = new hipEvent_t[realStreamNum];
	int h_prior;
	hipDeviceGetStreamPriorityRange(nullptr,&h_prior);
	for(int i = 0; i <realStreamNum; ++i)
	{
		hipStreamCreateWithPriority(&streams[i],hipStreamDefault,h_prior++);
		cudaCheckError();
		hipEventCreate(&cpyEvents[i]);
		cudaCheckError();
	}
	dim3 thsPerBlck(BATCH_W,BATCH_H);

	uchar* d_ptr = dev_mem;
	uchar* h_ptr = pitchedHMem;
	int streamsEnded = 0;
	for(int i = 0,ch_i = 0; i <realStreamNum; ++i,++ch_i)
	{
		ulong offsetH = ch_i * streamSize;
		ulong curStremSize = streamSize;
		uint curHeight = streamHeight;
		if(offsetH > realSizeCh)
		{
			ch_i = 0;
			d_ptr += realSizeCh;
			h_ptr += realSizeCh;
			offsetH = 0;
		}
		if(offsetH + streamSize > realSizeCh)
		{
			curStremSize = realSizeCh - offsetH;
			curHeight = curStremSize / (width * sizeof(uchar));
		}
		ulong offsetD = offsetH * 2;

		if(d_ptr + offsetD + 2 *  curStremSize > dev_mem + sizetoDevMAlloc) // if we exceeded gpu mem we back to the begin
		{
			d_ptr = dev_mem;
			offsetD = 0;
			for(int j = streamsEnded; j <= i; ++j)
			{
				hipEventSynchronize(cpyEvents[j]); // we have to wait for the streams to get memory
				cudaCheckError();
				streamsEnded++;
			}
		}

		dim3 blckNum(width/(BATCH_W * PXL_PER_THD) + 1,curHeight/(BATCH_H) + 1);


		/*printf("Stream%d  Host off: %d Dev off: %p curHeight: %d curStreamS: %d\n",i,
								(offsetH)/(kb*kb),
								(void *)(d_ptr +offsetD),
								curHeight,
								curStremSize/(kb*kb));*/

		hipMemcpyAsync(&d_ptr[offsetD], &h_ptr[offsetH], curStremSize, hipMemcpyHostToDevice, streams[i]);
		cudaCheckError();
		kernel<<<blckNum, thsPerBlck, 0, streams[i]>>>(&d_ptr[offsetD],&d_ptr[offsetD + curStremSize],width,curHeight,PXL_PER_THD,IMAGE_SCALE);
		cudaCheckError();
		hipMemcpyAsync(&h_ptr[offsetH], &d_ptr[offsetD + curStremSize], curStremSize, hipMemcpyDeviceToHost, streams[i]);
		hipEventRecord(cpyEvents[i], streams[i]);
		cudaCheckError();
	}
	hipDeviceSynchronize();
	for(int i = 0; i <realStreamNum; ++i)
	{
		hipStreamDestroy(streams[i]); // we have to free the stream resources for the others
		cudaCheckError();
		hipEventDestroy(cpyEvents[i]);
		cudaCheckError();
	}
	testRead(pitchedHMem,realSize);

	hipFree(dev_mem);
	cudaCheckError();
	hipHostFree(pitchedHMem);
	cudaCheckError();
	Timer::getInstance().stop(name);
}

void testOversubUM(kernelPtr kernel)
{
	const ulong kb = 1024;
	const ulong imgSize = kb * kb * kb * IMG_SIZE_GB;
	const uint channelNum = 3;
	uint width = sqrt(imgSize / (channelNum * sizeof(uchar)));
	uint height = width;
	ulong realSizeCh = width * height * sizeof(uchar);
	ulong realSize = realSizeCh * channelNum;
	const std::string name = "Oversubscription unified memory";
	Timer::getInstance().start(name);
	int device =-1;
	hipGetDevice(&device);
	uchar* um_data = createUMem(nullptr,realSize);
	memset(um_data, 0, realSize);
	hipMemAdvise(um_data,realSize,hipMemAdviseSetReadMostly,device);
	cudaCheckError();
	hipMemPrefetchAsync(um_data,realSize,device,NULL);
	cudaCheckError();
	exec_kernel(um_data,um_data + realSize,width,height,kernel);
	hipDeviceSynchronize();
	cudaCheckError();
	hipMemPrefetchAsync(um_data + realSize,realSize,hipCpuDeviceId,NULL);
	cudaCheckError();
	testRead(um_data + realSize,realSize);

	hipFree(um_data);
	cudaCheckError();
	Timer::getInstance().stop(name);
}


void testStreamImgProcessingStd(kernelPtr kernel)
{
	std::string name = "Stream Image Processing Std";
    Timer::getInstance().start(name);
	const uint size = IMG_H * IMG_W * 3;
	const uint size_bytes = size * sizeof(uchar);
	const int streamNum = 3;
	const ulong totalSize = sizeof(uchar) * IMG_NUM * size;
	hipStream_t streams[streamNum];
	uchar* d_mem = createStdMem(nullptr,totalSize);
	uchar* h_mem = new uchar[totalSize];
	memset(h_mem, 0, totalSize);
	uchar* c_res_mem = d_mem + size * IMG_NUM;

	for(int i = 0; i <streamNum; ++i)
	{
		hipStreamCreate(&streams[i]);
		cudaCheckError();
	}

	dim3 thsPerBlck(BATCH_W,BATCH_H);
	dim3 blckNum(IMG_W/(BATCH_W * PXL_PER_THD) + 1,IMG_H/(BATCH_H) + 1);

	for(int i =0; i< IMG_NUM;++i)
	{
		int s_id = i % streamNum;
		 hipMemcpyAsync(&d_mem[i], &h_mem[i],size_bytes,
		                 hipMemcpyHostToDevice, streams[s_id]);
	}


	for(int i =0; i< IMG_NUM;++i)
	{
		int s_id = i % streamNum;
		kernel<<<blckNum, thsPerBlck, 0, streams[s_id]>>>(d_mem + i * size_bytes,c_res_mem + i * size_bytes,
														  IMG_W,IMG_H,PXL_PER_THD,IMAGE_SCALE);
	}

	for(int i =0; i< IMG_NUM;++i)
	{
		int s_id = i % streamNum;
		 hipMemcpyAsync(&h_mem[i], &d_mem[i],size_bytes,
				 	 	 hipMemcpyDeviceToHost, streams[s_id]);
	}
	hipDeviceSynchronize();
	cudaCheckError();
	testRead(h_mem,totalSize);
    Timer::getInstance().stop(name);
	hipFree(d_mem);
	delete h_mem;
}

void testStreamImgProcessingUm(kernelPtr kernel,bool withAdvise)
{
	std::string name = "Stream Image Processing UM";
	if(withAdvise)
		name += " Opt";
    Timer::getInstance().start(name);

	const ulong size = IMG_H * IMG_W * 3;
	const ulong totalSize = sizeof(uchar) * IMG_NUM * size;
	const int streamNum = 3;
	uchar* umem = createUMem(nullptr,totalSize);
	uchar* res_umem = umem + size * IMG_NUM;
	memset(umem, 0, totalSize);
	hipStream_t streams[streamNum];
	hipEvent_t events[streamNum];
	for(int i = 0; i <streamNum; ++i)
	{
		hipStreamCreate(&streams[i]);
		cudaCheckError();
		hipEventCreate(&events[i]);
		cudaCheckError();
	}

	dim3 thsPerBlck(BATCH_W,BATCH_H);
	dim3 blckNum(IMG_W/(BATCH_W * PXL_PER_THD) + 1,IMG_H/(BATCH_H) + 1);

	int device =-1;
	hipGetDevice(&device);
	cudaCheckError();
	if(withAdvise)
	{
		hipMemPrefetchAsync(umem,sizeof(uchar) * size, device,streams[1]);
		cudaCheckError();
		hipEventRecord(events[0], streams[1]);
		cudaCheckError();
	}
	for(int i =0; i< IMG_NUM;++i)
	{
		int s_id = i % streamNum;
		int s1_id = (i + 1) % streamNum;
		hipEventSynchronize(events[s_id]);
		hipEventSynchronize(events[s1_id]);
		kernel<<<blckNum, thsPerBlck, 0, streams[s_id]>>>(umem + i * size,res_umem + i * sizeof(uchar) *size,
														  IMG_W,IMG_H,PXL_PER_THD,IMAGE_SCALE);
		hipEventRecord(events[s_id], streams[s_id]);

		if(withAdvise)
		{
			if(i < IMG_NUM -1)
			{
				hipStreamSynchronize(streams[s1_id]);
				hipMemPrefetchAsync(umem + (i+1) * size, sizeof(uchar) * size, device, streams[s1_id]);
				hipEventRecord(events[s1_id], streams[s1_id]);
			}
			hipMemPrefetchAsync(res_umem + i * size, sizeof(uchar) * size, hipCpuDeviceId, streams[s_id]);
		}
	}
	hipDeviceSynchronize();
	cudaCheckError();
	testRead(res_umem,totalSize);
    Timer::getInstance().stop(name);
	hipFree(umem);
}

void testFluidSimStd()
{
    Timer::getInstance().start("Fluid simulation std mem");
    StartArgs args = parsInputArguments();
    Fraction* space = initSpace(args,false,false);
    FluidParams *d_params, params = initParams();

    hipMalloc((void **)&d_params, sizeof(FluidParams));
    hipMemcpy(d_params, &params, sizeof(FluidParams), hipMemcpyHostToDevice);

    if (NULL == space)
        exit(-1);

    Fraction *d_space, *d_result;
    int totalSize = sizeof(Fraction)*args.SIZE(), i;
    void *result = new Fraction[totalSize];

    hipMalloc((void **)&d_space, totalSize);
    hipMalloc((void **)&d_result, totalSize);
    cudaCheckErrors("Mallocs");
    hipMemcpy(d_space, space, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, totalSize, hipMemcpyHostToDevice);

    cudaCheckErrors("Copy mem");
    printf("StdMem Simulation started\n");

    for (i = 0; i<args.NUM_OF_ITERATIONS; ++i)
    {
        simulation(args, d_params, d_space, d_result);
        swapPointers(d_space, d_result);
        hipDeviceSynchronize();

    	if (i % 2 == 0)
    		hipMemcpy(space, d_space, totalSize, hipMemcpyDeviceToHost);
    	else
    		hipMemcpy(space, d_result, totalSize, hipMemcpyDeviceToHost);

    	testRead(space,totalSize);
    }


    Timer::getInstance().stop("Fluid simulation std mem");
    printf("Simulation completed\n");
    free(space);
    hipFree(d_params);
    hipFree(d_space);
    hipFree(d_result);
    cudaCheckErrors("Free mem");
}

void testFluidSimUM(bool withAdvise)
{
    std::string name = withAdvise ? "Fluid simulation UM advised" : "Fluid simulation UM std";
    Timer::getInstance().start(name);

	int device = -1;
	hipGetDevice(&device);
    StartArgs args = parsInputArguments();
    int totalSize = sizeof(Fraction)*args.SIZE(), i;
    FluidParams *um_params, params = initParams();
    hipMallocManaged(&um_params, sizeof(FluidParams));
    memcpy(um_params,&params,sizeof(FluidParams));
    Fraction *buffer,*result,*start;
    hipMallocManaged(&buffer,totalSize);
    cudaCheckErrors("UM Mallocs");
    if(withAdvise)
    {
    	printf("UM advised Simulation started\n");
    	hipMemAdvise(um_params,sizeof(FluidParams),hipMemAdviseSetReadMostly,device);
    	cudaCheckError();
    	hipMemPrefetchAsync(um_params,sizeof(FluidParams),device,NULL);
    	cudaCheckError();
    	hipMemPrefetchAsync(buffer,totalSize,device,NULL);
    	cudaCheckError();
    }
    else
    {
    	printf("UM Simulation started\n");
    }
    Fraction* space = initSpace(args,true,withAdvise,device);
    for (i = 0; i<args.NUM_OF_ITERATIONS; ++i)
    {
        simulation(args, um_params, space, buffer);
        swapPointers(space, buffer);
        hipDeviceSynchronize();

    	result = (i % 2 == 0) ? space : buffer;
    	start = (i % 2 == 0)  ? buffer : space;

        if(withAdvise)
        {
        	hipMemPrefetchAsync(result,totalSize,hipCpuDeviceId,NULL);
        	cudaCheckError();
        	hipMemPrefetchAsync(start,totalSize,device,NULL);
        	cudaCheckError();
        }

    	testRead(result,totalSize);

        if(withAdvise)
        {
        	hipMemPrefetchAsync(result,totalSize,device,NULL);
        	cudaCheckError();
        }
    }

    Timer::getInstance().stop(name);
    printf("Simulation completed\n");

    hipFree(space);
    hipFree(buffer);
    hipFree(um_params);
}

