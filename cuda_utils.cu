#include "headers.h"
#include "config.h"

void initCuda()
{
	int nDevices;
    int devCount = hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        cudaCheckError()
        printf("CUDA device [%s] has %d Multi-Processors\n",
               props.name, props.multiProcessorCount);
    }
}

void exec_kernel(uchar* in, uchar* out,size_t width, size_t height,kernel kernel_ptr)
{
	dim3 thsPerBlck(BATCH_W,BATCH_H);
	dim3 blckNum(width/(BATCH_W * PXL_PER_THD) + 1,height/(BATCH_H) + 1);
	auto in_ptr = in;
	auto out_ptr = out;
	auto ch_size = sizeof(uchar) * width * height;
	for(int i = 0; i < CHANNEL_NUM;++i)
	{
		kernel_ptr<<<blckNum, thsPerBlck>>>(in_ptr,out_ptr,(uint)width,(uint)height,PXL_PER_THD,IMAGE_SCALE);
		in_ptr += ch_size;
		out_ptr += ch_size;
	}
}
