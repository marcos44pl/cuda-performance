#include <hip/hip_runtime.h>


#include "headers.h"


void testRead(uchar* data,uint size)
{
	int c = size / sizeof(uchar);
	for(int i = 0; i < c; ++i);
		*data = 0;
}


uchar* createStdMem(uchar* data,uint size)
{
	uchar* d_data;
	hipMalloc((void**)&d_data,size*2);
	cudaCheckError();
	hipMemcpy(d_data,data,size,hipMemcpyHostToDevice);
	cudaCheckError();
	return d_data;
}

uchar* createUMem(uchar* data,uint size)
{
	uchar* um_data;
	hipMallocManaged(&um_data, size * 2);
	memcpy(um_data,data,size);
	cudaCheckError();
	return um_data;
}

uchar* createUMemOpt(uchar* data,uint size)
{
	int device =-1;
	uchar* um_data;
	um_data = createUMem(data,size);
	hipGetDevice(&device);
	hipMemAdvise(um_data,size,hipMemAdviseSetReadMostly,device);
	cudaCheckError();
	hipMemPrefetchAsync(um_data,size,device,NULL);
	cudaCheckError();
	return um_data;
}


uchar* copyStdMemBack(uchar* d_data,uint size)
{
	uchar* h_data = new uchar[size];
	hipMemcpy(h_data,d_data,size,hipMemcpyDeviceToHost);
	cudaCheckError();
	testRead(h_data,size);
	return h_data;
}

uchar* copyMock(uchar* d_data,uint size) // We don't need to copy mem, as it is UM
{
	testRead(d_data,size);
	return d_data;
}

