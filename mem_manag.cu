#include <hip/hip_runtime.h>


#include "headers.h"
#include "utils.h"

uchar* createStdMem(uchar* data,ulong size)
{
	uchar* d_data;
	hipMalloc((void**)&d_data,size*2);
	cudaCheckError();
	if(data)
		hipMemcpy(d_data,data,size,hipMemcpyHostToDevice);
	cudaCheckError();
	return d_data;
}

uchar* createUMem(uchar* data,ulong size)
{
	uchar* um_data;
	hipMallocManaged(&um_data, size * 2);
	cudaCheckError();
	if(data)
		memcpy(um_data,data,size);
	return um_data;
}

uchar* createUMemOpt(uchar* data,ulong size)
{
	int device =-1;
	uchar* um_data;
	um_data = createUMem(data,size);
	hipGetDevice(&device);
	cudaCheckError();
	hipMemAdvise(um_data,size,hipMemAdviseSetReadMostly,device);
	cudaCheckError();
	hipMemPrefetchAsync(um_data,size,device,NULL);
	cudaCheckError();
	return um_data;
}


uchar* copyStdMemBack(uchar* d_data,ulong size)
{
	uchar* h_data = new uchar[size];
	if (h_data == nullptr)
		printf("Out of memory\n");
	hipMemcpy(h_data,d_data,size,hipMemcpyDeviceToHost);
	cudaCheckError();
	testRead(h_data,size);
	return h_data;
}

uchar* copyMock(uchar* d_data,ulong size) // We don't need to copy mem, as it is UM
{
	testRead(d_data,size);
	return d_data;
}

void freeStd(uchar* d_in,uchar* d_out,uchar* h_out)
{
	hipFree(d_in);
	cudaCheckError();
	delete h_out;
}


void freeUM(uchar* d_in,uchar* d_out,uchar* h_out)
{
	hipFree(d_in);
	cudaCheckError();
}

